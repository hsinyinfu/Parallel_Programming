#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>

#define MAXPOINTS 1000000
#define MAXSTEPS 1000000
#define MINPOINTS 20
#define PI 3.14159265

void check_param(void);
void init_line(void);
void update (void);
void printfinal (void);
__device__ void do_math(float*, float*, float*, int );

//---------------------------------------------------------------------------
//	Kernel function
//---------------------------------------------------------------------------
__global__ void gpu_init_line( float*, float*, int );
__global__ void gpu_update( float*, float*, float*, int, int );


int nsteps,                 	/* number of time steps */
	tpoints, 	     		/* total points along string */
	rcode;                  	/* generic return code */

float  values[MAXPOINTS+2], 	/* values at time t */
	   oldval[MAXPOINTS+2], 	/* values at time (t-dt) */
	   newval[MAXPOINTS+2]; 	/* values at time (t+dt) */

float *gValues, 
	  *gOldVal, 
	  *gNewVal;


/**********************************************************************
 *	Checks input values from parameters
 *********************************************************************/
void check_param(void)
{
	char tchar[20];

	/* check number of points, number of iterations */
	while ((tpoints < MINPOINTS) || (tpoints > MAXPOINTS)) {
		printf("Enter number of points along vibrating string [%d-%d]: "
				,MINPOINTS, MAXPOINTS);
		scanf("%s", tchar);
		tpoints = atoi(tchar);
		if ((tpoints < MINPOINTS) || (tpoints > MAXPOINTS))
			printf("Invalid. Please enter value between %d and %d\n", 
					MINPOINTS, MAXPOINTS);
	}
	while ((nsteps < 1) || (nsteps > MAXSTEPS)) {
		printf("Enter number of time steps [1-%d]: ", MAXSTEPS);
		scanf("%s", tchar);
		nsteps = atoi(tchar);
		if ((nsteps < 1) || (nsteps > MAXSTEPS))
			printf("Invalid. Please enter value between 1 and %d\n", MAXSTEPS);
	}

	printf("Using points = %d, steps = %d\n", tpoints, nsteps);

}

/**********************************************************************
 *     Initialize points on line
 *********************************************************************/
__global__ void gpu_init_line( float *gOld, float *gVal, int tpoints )
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;

	/* Initialize old values array */
	gOld[index] = gVal[index];
}

void init_line(void)
{
	//int i, j;
	int j;
	float x, fac, k, tmp;

	/* Calculate initial values based on sine curve */
	fac = 2.0 * PI;
	k = 0.0; 
	tmp = tpoints - 1;
	for (j = 1; j <= tpoints; j++) {
		x = k/tmp;
		values[j] = sin (fac * x);
		k = k + 1.0;
	} 

	hipMemcpy( gValues, values, (MAXPOINTS+2)*sizeof(float),
			hipMemcpyHostToDevice );

	dim3 dimBlock( 512 ), dimGrid( (tpoints + dimBlock.x -1) / dimBlock.x );
	gpu_init_line<<<dimGrid, dimBlock>>>( gOldVal, gValues, tpoints );
}


/**********************************************************************
 *      Calculate new values using wave equation
 *********************************************************************/
__device__ void do_math(float *gOld, float *gVal, float *gNew, int i)
{
	float dtime, c, dx, tau, sqtau;

	dtime = 0.3;
	c = 1.0;
	dx = 1.0;
	tau = (c * dtime / dx);
	sqtau = tau * tau;
	gNew[i] = (2.0 * gVal[i]) - gOld[i] + (sqtau *  (-2.0)*gVal[i]);
}

/**********************************************************************
 *     Update all values along line a specified number of times
 *********************************************************************/
__global__ void gpu_update( float *gOld, float *gVal, float *gNew, int nsteps,
		int tpoints )
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;

	/* Update values for each time step */
	for( int i=0; i < nsteps; i++ ) {
		/* global endpoints */
		if( (index == 1) || (index == tpoints) )
			gNew[index] = 0.0;
		else
			do_math( gOld, gVal, gNew, index );

		/* Update old values with new values */
		gOld[index] = gVal[index];
		gVal[index] = gNew[index];
	}
}

void update()
{

	dim3 threadsPerBlock( 512 ),
		 blocksPerGrid( (tpoints + threadsPerBlock.x -1) / threadsPerBlock.x );
	gpu_update<<<blocksPerGrid, threadsPerBlock>>>( gOldVal, gValues, gNewVal, nsteps, tpoints );

	hipMemcpy( values, gValues, (MAXPOINTS+2)*sizeof(float),
			hipMemcpyDeviceToHost );
}


/**********************************************************************
 *     Print final results
 *********************************************************************/
void printfinal()
{
	int i;

	for (i = 1; i <= tpoints; i++) {
		printf("%6.4f ", values[i]);
		if (i%10 == 0)
			printf("\n");
	}
}

/**********************************************************************
 *	Main program
 *********************************************************************/
int main(int argc, char *argv[])
{
	sscanf(argv[1],"%d",&tpoints);
	sscanf(argv[2],"%d",&nsteps);

	/*	Allocate space in device Global Memory	*/
	hipMalloc( (void**)&gValues, (MAXPOINTS+2)*sizeof(float) );
	hipMalloc( (void**)&gOldVal, (MAXPOINTS+2)*sizeof(float) );
	hipMalloc( (void**)&gNewVal, (MAXPOINTS+2)*sizeof(float) );

	check_param();
	printf("Initializing points on the line...\n");
	init_line();
	printf("Updating all points for all time steps...\n");
	update();
	printf("Printing final results...\n");
	printfinal();
	printf("\nDone.\n\n");

	/*	Free allocated space in device Global Memory	*/
	hipFree( gValues );
	hipFree( gOldVal );
	hipFree( gNewVal );

	return 0;
}
